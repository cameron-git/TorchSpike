#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z)
{
    return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z)
{
    const auto s = sigmoid(z);
    return (1.0 - s) * s;
}

template <typename scalar_t>
__global__ void lif_cuda_forward_kernel(
    scalar_t *__restrict__ x,
    scalar_t *__restrict__ v,
    scalar_t *__restrict__ vh,
    size_t neuron_size,
    float th,
    float tau)
{
    const int column = blockIdx.x * blockDim.x + threadIdx.x;
    const int index = blockIdx.y * neuron_size + column;
    if (column < neuron_size)
    {
        vh[index] = v[index] + (x[index] - v[index]) / tau;
        x[index] = vh[index] >= th;
        v[index] = vh[index] * (1.0 - x[index]);
    }
}
std::vector<at::Tensor> lif_cuda_forward(
    torch::Tensor x,
    torch::Tensor v,
    float th,
    float tau)
{
    const auto batch_size = x.size(0);
    const auto neuron_size = x.size(1);

    auto vh = torch::zeros_like(v);

    const int threads = 1024;
    // threads - 1 is to make sure we always have a fraction of a block more than needed
    const dim3 blocks((neuron_size + threads - 1) / threads, batch_size);

    AT_DISPATCH_FLOATING_TYPES(
        x.type(),
        "lif_forward_cuda",
        ([&]
         { lif_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
               x.data<scalar_t>(),
               v.data<scalar_t>(),
               vh.data<scalar_t>(),
               neuron_size,
               th,
               tau); }));
    return {x, v, vh};
}

template <typename scalar_t>
__global__ void lif_cuda_backward_kernel(
    scalar_t *__restrict__ grad_x,
    scalar_t *__restrict__ grad_v,
    scalar_t *__restrict__ x,
    scalar_t *__restrict__ vh,
    size_t neuron_size,
    float th,
    float tau)
{
    const int column = blockIdx.x * blockDim.x + threadIdx.x;
    const int index = blockIdx.y * neuron_size + column;
    if (column < neuron_size)
    {
        grad_x[index] = grad_x[index] + grad_v[index] * -vh[index];
        grad_v[index] = grad_v[index] * (1 - x[index]) + grad_x[index] * d_sigmoid(vh[index] - th);
        grad_x[index] = grad_v[index] * (1 / tau);
        grad_v[index] = grad_v[index] * (1 - 1 / tau);
    }
}

std::vector<at::Tensor> lif_cuda_backward(
    torch::Tensor grad_x,
    torch::Tensor grad_v,
    torch::Tensor x,
    torch::Tensor vh,
    float th,
    float tau)
{
    const auto batch_size = x.size(0);
    const auto neuron_size = x.size(1);

    const int threads = 1024;
    // threads - 1 is to make sure we always have a fraction of a block more than needed
    const dim3 blocks((neuron_size + threads - 1) / threads, batch_size);

    AT_DISPATCH_FLOATING_TYPES(
        x.type(),
        "lif_backward_cuda",
        ([&]
         { lif_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
               grad_x.data<scalar_t>(),
               grad_v.data<scalar_t>(),
               x.data<scalar_t>(),
               vh.data<scalar_t>(),
               neuron_size,
               th,
               tau); }));

    return {grad_x, grad_v};
}
